#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x;
    c[index] = a[index] + b[index];
}

int main() {
    int size = 5;
    int h_a[size] = {1, 2, 3, 4, 5};
    int h_b[size] = {10, 20, 30, 40, 50};
    int h_c[size]; // Host array to store result

    int *d_a, *d_b, *d_c; // Device pointers

    // Allocate memory on the device
    hipMalloc((void **)&d_a, size * sizeof(int));
    hipMalloc((void **)&d_b, size * sizeof(int));
    hipMalloc((void **)&d_c, size * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with one block of size threads
    add<<<1, size>>>(d_a, d_b, d_c);

    // Copy result back from device to host
    hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < size; i++) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    hipDeviceSynchronize();

    return 0;
}